#include "hip/hip_runtime.h"
//
// Created by tsv on 01.03.17.
//

#include "burn_solver.hpp"
#include "cuda_utils.cuh"

__global__
void
kernel_step(float* g_in, float* g_out, size_t N, float h, float tau, CoercedParams params)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i > 0 && i < (N - 1)) {
		float dg0 = (g_in[1] - g_in[0]) / h;
		float gl = g_in[i - 1];
		float g = g_in[i];
		float gr = g_in[i + 1];

		float RT = params.RT(g);

		g_out[i] = (
						   RT / params.P *
						   (
								   params.D * (gr - 2 * g + gl) / (h * h)
								   - (params.B * dg0 + params.Q) / params.C * (gr - gl) / (2.f * h)
						   )
						   + (1.f - g) * params.A * expf(params.E / RT)
				   ) * tau
				   + g;
	}

	if (i == (N - 1)) {
		g_out[i] = g_out[i - 1];
	}
}

void
BurnSolver::next_step()
{
	kernel_step<<<get_grids(), get_blocks()>>>(d_y_in, d_y_out, get_n(), get_h(), get_tau(), params.coerced_params());
	cuda_check_error(hipGetLastError());
	hipDeviceSynchronize();
}
